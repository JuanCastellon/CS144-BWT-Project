
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include<iterator>
#include<algorithm>
#include <vector>

std::vector<std::pair<char, int>> createRankList(std::string inputStr)
{
    
    std::vector<std::pair<char, int>> rank_lst;
    int a = 0;
    int c = 0;
    int g = 0;
    int t = 0;
    int d = 0;

    for (auto& s : inputStr)
    {
        char letter;
        int pos;
        switch (s)
        {
        case '$':
            d += 1;
            letter = s;
            pos = d;
            break;

        case 'A':
            a += 1;
            letter = s;
            pos = a;
            break;

        case 'C':
            c += 1;
            letter = s;
            pos = c;

            break;

        case 'G':
            g += 1;
            letter = s;
            pos = g;

            break;

        case 'T':
            t += 1;
            letter = s;
            pos = t;

            break;
        default:
            std::cout << " no case was chosen\n";
        }
        std::pair <char, int> rankPair = std::make_pair(letter,pos);
        rank_lst.push_back(rankPair);
    }
    return rank_lst;
}


std::vector<std::pair<char, int>> createHelperList(std::string inputStr)
{
    std::vector<std::pair<char, int> > helper_lst;
    std::sort(inputStr.begin(), inputStr.end());
    std::reverse(inputStr.begin(), inputStr.end());
    int a = 0;
    int c = 0;
    int g = 0;
    int t = 0;
    int d = 0;

    for (auto& s : inputStr)
    {
        char letter;
        int pos;
        switch (s)
        {
        case '$':
            d += 1;
            letter = s;
            pos = d;
            break;

        case 'A':
            a += 1;
            letter = s;
            pos = a;
            break;

        case 'C':
            c += 1;
            letter = s;
            pos = c;

            break;

        case 'G':
            g += 1;
            letter = s;
            pos = g;

            break;

        case 'T':
            t += 1;
            letter = s;
            pos = t;

            break;
        default:
            std::cout << " no case was chosen\n";
        }
        std::pair <char, int> rankPair = std::make_pair(letter, pos);
        helper_lst.push_back(rankPair);
      
    }
    return helper_lst;
}

void computeSuffixArray(std::string input_text)
{
    std::vector< std::pair< std::string, int> > suffixArr; // list of tuple < string text_input, index>

    int index = 0;

    for (int i = 0; i < input_text.length(); i++)
    {
        std::cout << input_text << std::endl;
        std::pair <std::string, int> suffPair = std::make_pair(input_text, index);
        suffixArr.push_back(suffPair);
  
        std::rotate(input_text.rbegin(), input_text.rbegin() + 1, input_text.rend());
        index++;

    }

    for (auto& x : suffixArr)
    {
        std::cout << " (" << x.first << "[" << x.first[0] << "]" << "," << x.second << ")";
    }
    std::cout << "we did it bois before sort print " << std::endl;
    sort(suffixArr.begin(), suffixArr.end());
   

    for (auto& x : suffixArr)
    {
        std::cout << " (" << x.first << "[" << x.first[0] << "]" << "," << x.second << ")";
    }
    std::cout << "we did it bois after sort" << std::endl;

}

std::vector<std::string> rotation(std::string input_text) {
    std::vector<std::string> rotations;
    std::string first;

    for (int i = 0; i < input_text.length(); i++) {
        input_text = input_text.substr(1, input_text.length() - 1) + input_text.front();
        rotations.push_back(input_text);
    }
    std::cout << "==========" << std::endl;
    sort(rotations.begin(), rotations.end());
 
    for (auto const& i : rotations) {
        std::cout << i << std::endl;
    }
    return rotations;
}


void bwt(std::string input_text) {


    std::vector<std::string> sortedlist = rotation(input_text);
    std::vector<std::string> finalList;
    std::string first;

    std::vector<std::string>::iterator it;

    std::string finalString;
    for (auto& x : sortedlist)
    {

        finalString.push_back(x.back());

        std::cout << "this is finalString: " << finalString << std::endl;
    }
    std::cout << "==========" << std::endl;
    sort(finalList.begin(), finalList.end());

    for (auto const& i : finalList) {
        std::cout << i << std::endl;
    }

}

int main()
{
    std::string inputStr = "ACTGGCT$TGCGGC";

    std::vector<std::pair<char, int> > rank_lst = createRankList(inputStr);
    std::vector<std::pair<char, int> > helper_lst = createHelperList(inputStr);

    for (auto& x : rank_lst)
    {
        std::cout << " (" << x.first << "," << x.second << ")";
    }
    std::cout << "we did it bois " << std::endl;


    for (auto& x : helper_lst)
    {
        std::cout << " (" << x.first << "," << x.second << ")";
    }
    std::cout << "we did it bois " << std::endl;

    computeSuffixArray("banana$");




    bwt("ACTGGCT$TGCGGC");

    return 0;
}
